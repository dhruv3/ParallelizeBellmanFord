#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "utils.h"
#include "cuda_error_check.cuh"
#include "initial_graph.hpp"
#include "parse_graph.hpp"

__global__ void neighborHandling_kernel(std::vector<initial_vertex> * peeps, int offset, int * anyChange){

    //update me based on my neighbors. Toggle anyChange as needed.
    //Enqueue and dequeue me as needed.
    //Offset will tell you who I am.
}

void puller_incore_impl2(std::vector<initial_vertex> * graph, int blockSize, int blockNum, ofstream &outputFile){
	unsigned int *initDist, *distance_cur, *distance_prev; 
	int *anyChange;
	//todo
	int *hostAnyChange = (int*)malloc(sizeof(int));
	graph_node *edge_list, *L;
	unsigned int edge_counter;
	edge_counter = total_edges(*graph);
	edge_list = (graph_node*) malloc(sizeof(graph_node)*edge_counter);
	
	//TODO: calloc changed to malloc
	initDist = (unsigned int*)malloc(sizeof(unsigned int)*graph->size());	
	initDist[0] = 0;
	for(int i = 1; i < graph->size(); i++){
	    initDist[i] = UINT_MAX; 
	}
	//set_edges(*graph, edge_list, edge_counter);
	unsigned int k = 0;
	for(int i = 0 ; i < graph->size() ; i++){
		std::vector<neighbor> nbrs = (*graph)[i].nbrs;
	    for(int j = 0 ; j < nbrs.size() ; j++, k++){
			edge_list[k].src = nbrs[j].srcIndex;
			edge_list[k].dst = i;
			edge_list[k].weight = nbrs[j].edgeValue.weight;
	    }
	}

	//sort by source vertex
	//http://www.cplusplus.com/reference/cstdlib/qsort/
	qsort(edge_list, edge_counter, sizeof(graph_node), cmp_edge);			

	//todo
	unsigned int *hostDistanceCur = new unsigned int[graph->size()];

	hipMalloc((void**)&L, (size_t)sizeof(graph_node)*edge_counter);
	hipMalloc((void**)&distance_cur, (size_t)sizeof(unsigned int)*(graph->size()));
	hipMalloc((void**)&distance_prev, (size_t)sizeof(unsigned int)*(graph->size()));
	hipMalloc((void**)&anyChange, (size_t)sizeof(int));
	

	hipMemcpy(distance_cur, initDist, (size_t)sizeof(unsigned int)*(graph->size()), hipMemcpyHostToDevice);
	hipMemcpy(distance_prev, initDist, (size_t)sizeof(unsigned int)*(graph->size()), hipMemcpyHostToDevice);
	hipMemcpy(L, edge_list, (size_t)sizeof(graph_node)*edge_counter, hipMemcpyHostToDevice);
	
	hipMemset(anyChange, 0, (size_t)sizeof(int));

    setTime();

	for(int i=0; i < ((int) graph->size())-1; i++){
		edge_process<<<blockNum,blockSize>>>(L, edge_counter, distance_prev, distance_cur, anyChange);
		hipMemcpy(hostAnyChange, anyChange, sizeof(int), hipMemcpyDeviceToHost);
		if(!hostAnyChange[0]){
			break;
		} 
		else {
			hipMemset(anyChange, 0, (size_t)sizeof(int));
			hipMemcpy(hostDistanceCur, distance_cur, (sizeof(unsigned int))*(graph->size()), hipMemcpyDeviceToHost);
			hipMemcpy(distance_cur, distance_prev, (sizeof(unsigned int))*(graph->size()), hipMemcpyDeviceToDevice);
			hipMemcpy(distance_prev, hostDistanceCur,(sizeof(unsigned int))*(graph->size()), hipMemcpyHostToDevice);
		}
	}

	cout << "Took " << getTime() << "ms.\n";

	hipMemcpy(hostDistanceCur, distance_cur, (sizeof(unsigned int))*(graph->size()), hipMemcpyDeviceToHost);

	for(int i=0; i < graph->size(); i++){
		if(hostDistanceCur[i] == UINT_MAX){
		    outputFile << i << ":" << "INF" << endl;
		}
		else{
		    outputFile << i << ":" << hostDistanceCur[i] << endl; 
		}
	}

	hipFree(L);
	hipFree(distance_cur);
	hipFree(distance_prev);
	hipFree(anyChange);
	
	delete[] hostDistanceCur;
	free(initDist);
	free(edge_list);
}
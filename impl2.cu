#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "utils.h"
#include "cuda_error_check.cuh"
#include "initial_graph.hpp"
#include "parse_graph.hpp"

void puller_incore_impl2(std::vector<initial_vertex> * graph, int blockSize, int blockNum, ofstream &outputFile){
	
}

void puller_outcore_impl2(std::vector<initial_vertex> * graph, int blockSize, int blockNum, ofstream &outputFile){
	double t_filter, t_comp;
	t_filter = 0;
	t_comp = 0;

	unsigned int *initDist, *distance_cur, *distance_prev, *to_process_arr, *pred; 
	int *anyChange;
	int *checkIfChange = (int*)malloc(sizeof(int));
	graph_node *edge_list, *L, *T;
	unsigned int edge_counter, to_process_num;
	edge_counter = total_edges(*graph);
	edge_list = (graph_node*) malloc(sizeof(graph_node)*edge_counter);
	
	unsigned int *temp = (unsigned int*)malloc(sizeof(unsigned int));
	int total_threads = blockSize * blockNum;
	int warp_num;
	if(total_threads % 32 == 0){
		warp_num = total_threads/32;
	}
	else{
		warp_num = total_threads/32 + 1;
	}


	//set initial distance to max except for source node
	initDist = (unsigned int*)malloc(sizeof(unsigned int)*graph->size());	
	initDist[0] = 0;
	for(int i = 1; i < graph->size(); i++){
	    initDist[i] = UINT_MAX; 
	}
	
	//for each member of edge list set initial values
	unsigned int k = 0;
	for(int i = 0 ; i < graph->size() ; i++){
		std::vector<neighbor> nbrs = (*graph)[i].nbrs;
	    for(int j = 0 ; j < nbrs.size() ; j++, k++){
			edge_list[k].src = nbrs[j].srcIndex;
			edge_list[k].dst = i;
			edge_list[k].weight = nbrs[j].edgeValue.weight;
	    }
	}

	//sort by source vertex
	//http://www.cplusplus.com/reference/cstdlib/qsort/
	qsort(edge_list, edge_counter, sizeof(graph_node), cmp_edge);			

	unsigned int *swapDistVariable = new unsigned int[graph->size()];
	unsigned int *hostTPA = new unsigned int[warp_num];

	hipMalloc((void**)&to_process_arr, (size_t)sizeof(unsigned int) * warp_num);
	hipMalloc((void**)&pred, (size_t)sizeof(unsigned int) * (graph->size()));
	hipMalloc((void**)&L, (size_t)sizeof(graph_node)*edge_counter);
	hipMalloc((void**)&distance_cur, (size_t)sizeof(unsigned int)*(graph->size()));
	hipMalloc((void**)&distance_prev, (size_t)sizeof(unsigned int)*(graph->size()));
	hipMalloc((void**)&anyChange, (size_t)sizeof(int));
	

	hipMemcpy(distance_cur, initDist, (size_t)sizeof(unsigned int)*(graph->size()), hipMemcpyHostToDevice);
	hipMemcpy(distance_prev, initDist, (size_t)sizeof(unsigned int)*(graph->size()), hipMemcpyHostToDevice);
	hipMemcpy(L, edge_list, (size_t)sizeof(graph_node)*edge_counter, hipMemcpyHostToDevice);
	hipMemset(to_process_arr, 0, (size_t)sizeof(unsigned int)*warp_num);
	hipMemset(pred, 0, (size_t)sizeof(unsigned int)*(graph->size()));
	hipMemset(anyChange, 0, (size_t)sizeof(int));

	for(int i=0; i < ((int) graph->size())-1; i++){
		setTime();
		if(i == 0){
		    edge_process<<<blockNum,blockSize>>>(L, edge_counter, distance_prev, distance_cur, anyChange, pred);
		    hipDeviceSynchronize();
		} 
		else {
		    hipMemset(pred, 0, (size_t)sizeof(unsigned int)*(graph->size()));
		    edge_process<<<blockNum,blockSize>>>(L, edge_counter, distance_prev, distance_cur, anyChange, pred);
		    hipDeviceSynchronize();
		    hipFree(T);
		}
		t_comp += getTime();
		hipMemcpy(checkIfChange, anyChange, sizeof(int), hipMemcpyDeviceToHost);
		if(!checkIfChange[0]){
			break;
		} 
		else {
			hipMemset(anyChange, 0, (size_t)sizeof(int));
			hipMemcpy(distance_prev, distance_cur, (sizeof(unsigned int))*(graph->size()), hipMemcpyDeviceToDevice);
			hipMemcpy(swapDistVariable, distance_cur, (sizeof(unsigned int))*(graph->size()), hipMemcpyDeviceToHost);
		}

		if(i == graph->size() - 2){
		    break;
		}
		else{
		    setTime();
		    hipMemset(to_process_arr, 0, (size_t)sizeof(unsigned int)*warp_num);
		    set_wrap_count<<<blockNum, blockSize>>>(L, edge_counter, pred, to_process_arr);
		    hipDeviceSynchronize();
		    hipMemcpy(temp, to_process_arr + warp_num - 1, sizeof(unsigned int), hipMemcpyDeviceToHost);
		    to_process_num = *temp;
		    hipMemcpy(hostTPA, to_process_arr, sizeof(unsigned int)*warp_num, hipMemcpyDeviceToHost);
		    thrust::exclusive_scan(hostTPA, hostTPA + warp_num, hostTPA);
		    hipMemcpy(to_process_arr, hostTPA, sizeof(unsigned int)*warp_num, hipMemcpyHostToDevice);
		    hipMemcpy(temp, to_process_arr + warp_num - 1, sizeof(unsigned int), hipMemcpyDeviceToHost);
		    to_process_num += *temp;
		    hipMalloc((void**)&T, (size_t)sizeof(edge_node)*to_process_num);
		    filter_T<<<blockNum, blockSize>>>(L, edge_counter, pred, to_process_arr, T);
		    hipDeviceSynchronize();
		    t_filter += getTime();
		}
	}

	printf("Computation Time: %f ms\nFiltering Time: %f ms\n", t_comp, t_filter);

	hipMemcpy(swapDistVariable, distance_cur, (sizeof(unsigned int))*(graph->size()), hipMemcpyDeviceToHost);

	for(int i=0; i < graph->size(); i++){
		if(swapDistVariable[i] == UINT_MAX){
		    outputFile << i << ":" << "INF" << endl;
		}
		else{
		    outputFile << i << ":" << swapDistVariable[i] << endl; 
		}
	}

	hipFree(L);
	hipFree(distance_cur);
	hipFree(distance_prev);
	hipFree(anyChange);

	delete[] hostTPA;
	delete[] swapDistVariable;
	free(initDist);
	free(edge_list);
}
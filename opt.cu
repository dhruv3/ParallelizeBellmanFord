#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <thrust/scan.h>
#include "utils.h"
#include "cuda_error_check.cuh"
#include "initial_graph.hpp"
#include "parse_graph.hpp"

using namespace std;

//comparator function used by qsort
int cmp_edge_src(const void *a, const void *b){
	return ( (((graph_node *)a)->src) - (((graph_node *)b)->src));
}

//get total edges
unsigned int total_edges_opt(std::vector<initial_vertex>& graph){
	unsigned int edge_counter = 0;
	for(int i = 0 ; i < graph.size() ; i++){
	    edge_counter += graph[i].nbrs.size();
	}
	return edge_counter;
}

__global__ void set_warp_count_opt(graph_node *L, unsigned int *warp_update_ds, const unsigned int edge_counter, unsigned int *flag){

  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int total_threads = blockDim.x * gridDim.x;
	int warp_id = thread_id/32;
	int warp_num;
	if(total_threads % 32 == 0){
		warp_num = total_threads/32;
	}
	else{
		warp_num = total_threads/32 + 1;
	}
	int lane_id = thread_id % 32;

    //given in the psuedocode
	int load = (edge_counter % warp_num == 0) ? edge_counter/warp_num : edge_counter/warp_num+1;
	int beg = load * warp_id;
	int end = beg + load;
	if(edge_counter < beg + load)
		end = edge_counter;
	beg = beg + lane_id;

    unsigned int temp_num = 0;
    graph_node *edge;
    for(int i = beg; i < end; i+=32){
    	edge = L + i;
    	//as per desc given
		int mask = __ballot(flag[edge->src]);
	    warp_update_ds[warp_id] += __popc(mask);
    }
}

__global__ void filter_T_opt(graph_node *L, uint *edge_offset_ds, unsigned int *warp_update_ds, const unsigned int edge_counter, unsigned int *flag){
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;
	int warp_id = thread_id/32;
	int warp_num;
	if(total_threads % 32 == 0){
		warp_num = total_threads/32;
	}
	else{
		warp_num = total_threads/32 + 1;
	}
	int lane_id = thread_id % 32;

    //given in the psuedocode
	int load = (edge_counter % warp_num == 0) ? edge_counter/warp_num : edge_counter/warp_num+1;
	int beg = load * warp_id;
	int end = beg + load;
	if(edge_counter < beg + load)
		end = edge_counter;
	beg = beg + lane_id;

	uint cur_offset = 0;
    graph_node *edge;
    for(int i = beg; i < end; i+=32){
    	edge = L + i;
    	//as per desc given
		int mask = __ballot(flag[L[i].src]);
		int inner_idx = __popc(mask << (32 - lane_id));
		if(flag[edge->src]){
		    edge_offset_ds[cur_offset + inner_idx + warp_update_ds[warp_id]]= i;
		}
		cur_offset += __popc(mask);
    }
}


//outcore
//kernel outcore method
//2 extra params:
//edge_offset_ds-
//flag- to set bit for the vertex whose src changes
__global__ void edge_process_opt(graph_node *L, uint *edge_offset_ds, const uint edge_counter, unsigned int *distance_cur, unsigned int *distance_prev, int *anyChange, unsigned int *flag){

	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int total_threads = blockDim.x * gridDim.x;
	int warp_id = thread_id/32;
	int warp_num;
	if(total_threads % 32 == 0){
		warp_num = total_threads/32;
	}
	else{
		warp_num = total_threads/32 + 1;
	}
	int lane_id = thread_id % 32;

	//given in the psuedocode
	int load = (edge_counter % warp_num == 0) ? edge_counter/warp_num : edge_counter/warp_num+1;
	int beg = load * warp_id;
	int end = beg + load;
	if(edge_counter < beg + load)
		end = edge_counter;
	beg = beg + lane_id;

	unsigned int u, v, w;
	graph_node *edge;
	for(int i = beg; i < end; i+=32){
		edge = L + edge_offset_ds[i];
		u = edge->src;
		v = edge->dst;
		w = edge->weight;
		if(distance_prev[u] != UINT_MAX && distance_prev[u] + w < distance_cur[v]){
			atomicMin(&distance_cur[v], distance_prev[u] + w);
			anyChange[0] = 1;
			flag[v] = 1;
		}
	}
}

//device outcore method
void puller_outcore_impl3(std::vector<initial_vertex> * graph, int blockSize, int blockNum, ofstream &outputFile){
	double filter_time = 0;
	double compute_time = 0;

	unsigned int *initDist;
	//set initial distance to max except for source node
	initDist = (unsigned int*)malloc(sizeof(unsigned int)*graph->size());
	initDist[0] = 0;
	for(int i = 1; i < graph->size(); i++){
	    initDist[i] = UINT_MAX;
	}

	unsigned int edge_counter = total_edges_opt(*graph);
	unsigned int initial_edge_counter = edge_counter;

	graph_node *edge_list;
	edge_list = (graph_node*) malloc(sizeof(graph_node)*edge_counter);
	//for each member of edge list set initial values
	unsigned int k = 0;
	for(int i = 0 ; i < graph->size() ; i++){
		std::vector<neighbor> nbrs = (*graph)[i].nbrs;
	    for(int j = 0 ; j < nbrs.size() ; j++, k++){
			edge_list[k].src = nbrs[j].srcIndex;
			edge_list[k].dst = i;
			edge_list[k].weight = nbrs[j].edgeValue.weight;
	    }
	}

	//sort by source vertex
	//http://www.cplusplus.com/reference/cstdlib/qsort/
	qsort(edge_list, edge_counter, sizeof(graph_node), cmp_edge_src);

	//create allneighbor
	unsigned int *allNeighborNumber = new unsigned int[graph->size()];
	for(int i = 0; i < graph->size(); i++){
		allNeighborNumber[i] = 0;
	}
	for(int i = 0; i < graph->size(); i++){
	    std::vector<neighbor> nbrs = (*graph)[i].nbrs;
	    for(int j = 0 ; j < nbrs.size() ; j++){
	    	int src = nbrs[j].srcIndex;
	    	allNeighborNumber[src] += 1;
	    }
	}

  //create allOffsets
  unsigned int *allOffsets = new unsigned int[graph->size() + 1];
  for(int i = 0; i < graph->size(); i++){
	    allOffsets[i] = allNeighborNumber[i];
	}
  thrust::exclusive_scan(allOffsets, allOffsets + graph->size(), allOffsets);
  allOffsets[graph->size()] = allOffsets[graph->size() - 1] + allNeighborNumber[graph->size() - 1];

	unsigned int vector_size = 0;
	//create a new l'
	graph_node *L_new;
	L_new = (graph_node*) malloc(sizeof(graph_node)*edge_counter);
	for(int i = 0, j=0 ; i < edge_counter; i++){
		if(edge_list[i].src == 0){
			L_new[j].src = edge_list[i].src;
			L_new[j].dst = edge_list[i].dst;
			L_new[j].weight = edge_list[i].weight;
			j++;
		}
		vector_size = j;
	}

	//create nodeQueue
	std::vector<int> nodeQueue(vector_size);
	std::vector<int> device_nodeQueue(vector_size);
	hipMalloc((void**)&device_nodeQueue, sizeof(unsigned int)*vector_size);
	//hipMemcpy(device_nodeQueue, &nodeQueue, sizeof(uint)*vector_size, hipMemcpyHostToDevice);

	//queueCounter
	unsigned int *queueCounter = 0;
	unsigned int *device_queueCounter;
	hipMalloc((void**)&device_queueCounter, sizeof(unsigned int));
	hipMemcpy(device_queueCounter, &queueCounter, sizeof(uint), hipMemcpyHostToDevice);
	hipMemset(device_queueCounter, 0, sizeof(unsigned int));

	unsigned int *distance_cur, *distance_prev, *flag, *temp_distance, *device_warp_update_ds, *device_edge_counter, *edge_offset_ds;
	graph_node *L;
	graph_node *device_L_new;
	int *anyChange, check_if_change;

	unsigned int *host_edge_offset_ds = new unsigned int[edge_counter];
	for (int i = 0; i < edge_counter; ++i) {
		host_edge_offset_ds[i] = i;
	}

	int total_threads = blockSize * blockNum;
	int warp_num;
	if(total_threads % 32 == 0){
		warp_num = total_threads/32;
	}
	else{
		warp_num = total_threads/32 + 1;
	}

	unsigned int *temp_warp_update = new unsigned int[warp_num];

	hipMalloc((void**)&edge_offset_ds, sizeof(unsigned int)*edge_counter);
	hipMalloc((void**)&device_edge_counter, sizeof(unsigned int));
	hipMalloc((void**)&temp_distance, sizeof(unsigned int)*graph->size());

	hipMalloc((void**)&device_warp_update_ds, sizeof(unsigned int)*warp_num);
	hipMalloc((void**)&flag, sizeof(unsigned int)*graph->size());
	hipMalloc((void**)&L, sizeof(graph_node)*edge_counter);
	hipMalloc((void**)&device_L_new, sizeof(graph_node)*edge_counter);
	hipMalloc((void**)&distance_cur, sizeof(unsigned int)*graph->size());
	hipMalloc((void**)&distance_prev, sizeof(unsigned int)*graph->size());
	hipMalloc((void**)&anyChange, sizeof(int));

	hipMemcpy(distance_cur, initDist, sizeof(unsigned int)*graph->size(), hipMemcpyHostToDevice);
	hipMemcpy(distance_prev, initDist, sizeof(unsigned int)*graph->size(), hipMemcpyHostToDevice);
	hipMemcpy(L, edge_list, sizeof(graph_node)*edge_counter, hipMemcpyHostToDevice);
	hipMemcpy(device_L_new, L_new, sizeof(graph_node)*edge_counter, hipMemcpyHostToDevice);
	hipMemcpy(device_edge_counter, &edge_counter, sizeof(uint), hipMemcpyHostToDevice);
	hipMemcpy(edge_offset_ds, host_edge_offset_ds, sizeof(uint)*edge_counter, hipMemcpyHostToDevice);

	for (int i = 0; i < graph->size()-1; ++i) {
		setTime();

		hipMemset(anyChange, 0, sizeof(int));
		hipMemset(flag, 0, sizeof(uint)*graph->size());
		hipMemset(device_warp_update_ds, 0, sizeof(uint)*warp_num);
		hipMemcpy(temp_distance, distance_cur, sizeof(uint)*graph->size(), hipMemcpyDeviceToDevice);

		edge_process_opt<<<blockNum, blockSize>>>(L, edge_offset_ds, initial_edge_counter, distance_cur, distance_prev, anyChange, flag);
		hipDeviceSynchronize();
		hipMemcpy(distance_prev, distance_cur, sizeof(uint)*graph->size(), hipMemcpyDeviceToDevice);
		hipMemcpy(&check_if_change, anyChange, sizeof(int), hipMemcpyDeviceToHost);

		compute_time += getTime();

		if (check_if_change == 0) {
			break;
		}

		setTime();

		set_warp_count_opt<<<blockNum, blockSize>>>(L, device_warp_update_ds, edge_counter, flag);
		hipDeviceSynchronize();

		hipMemcpy(temp_warp_update, device_warp_update_ds, sizeof(uint)*warp_num, hipMemcpyDeviceToHost);
		thrust::exclusive_scan(temp_warp_update, temp_warp_update + warp_num, temp_warp_update);
		hipDeviceSynchronize();
		hipMemcpy(device_warp_update_ds, temp_warp_update, sizeof(uint)*warp_num, hipMemcpyHostToDevice);

		filter_T_opt<<<blockNum, blockSize>>>(L, edge_offset_ds, device_warp_update_ds, edge_counter, flag);
		hipDeviceSynchronize();
		hipMemcpy(&initial_edge_counter, device_edge_counter, sizeof(uint), hipMemcpyDeviceToHost);
		hipMemcpy(host_edge_offset_ds, edge_offset_ds, sizeof(uint)*edge_counter, hipMemcpyDeviceToHost);

		filter_time += getTime();
	}

	std::cout << "Compute Time: " << compute_time << "\n";
	std::cout << "Filter Time: " << filter_time;

	hipMemcpy(initDist, distance_cur, sizeof(uint)*graph->size(), hipMemcpyDeviceToHost);

	for(int i=0; i < graph->size(); i++){
		if(initDist[i] == UINT_MAX){
		    outputFile << i << ":" << "INF" << endl;
		}
		else{
		    outputFile << i << ":" << initDist[i] << endl;
		}
	}
	free(initDist);
	delete[] host_edge_offset_ds;
	hipFree(L);
	hipFree(edge_offset_ds);
	hipFree(device_edge_counter);
	hipFree(flag);
	hipFree(distance_cur);
	hipFree(distance_prev);
	hipFree(temp_distance);
	hipFree(device_warp_update_ds);
	hipFree(anyChange);
}

#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <thrust/scan.h>
#include <algorithm>
#include "utils.h"
#include "cuda_error_check.cuh"
#include "initial_graph.hpp"
#include "parse_graph.hpp"

using namespace std;

//comparator function used by qsort
int cmp_edge_src(const void *a, const void *b){
	return ( (((graph_node *)a)->src) - (((graph_node *)b)->src));
}

//get total edges
unsigned int total_edges_opt(std::vector<initial_vertex>& graph){
	unsigned int edge_counter = 0;
	for(int i = 0 ; i < graph.size() ; i++){
	    edge_counter += graph[i].nbrs.size();
	}
	return edge_counter;
}

//outcore
//kernel outcore method
__global__ void edge_process_opt(graph_node *L, const uint edge_counter, unsigned int *distance_cur, unsigned int *distance_prev, unsigned int *queueCounter, unsigned int *nodeQueue){

	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int total_threads = blockDim.x * gridDim.x;
	int warp_id = thread_id/32;
	int warp_num;
	if(total_threads % 32 == 0){
		warp_num = total_threads/32;
	}
	else{
		warp_num = total_threads/32 + 1;
	}
	int lane_id = thread_id % 32;

	//given in the psuedocode
	int load = (edge_counter % warp_num == 0) ? edge_counter/warp_num : edge_counter/warp_num+1;
	int beg = load * warp_id;
	int end = beg + load;
	if(edge_counter < beg + load)
		end = edge_counter;
	beg = beg + lane_id;

	unsigned int u, v, w;
	graph_node *edge;
	for(int i = beg; i < end; i+=32){
		edge = L + i;
		u = edge->src;
		v = edge->dst;
		w = edge->weight;
		if(distance_prev[u] != UINT_MAX && distance_prev[u] + w < distance_prev[v] && distance_prev[u] + w < distance_cur[v]){
			int old_val = atomicMin(&distance_cur[v], distance_prev[u] + w);
			if(old_val >= distance_prev[v] && distance_prev[u] + w < old_val){
				int idx = atomicAdd(&queueCounter[0], 1);
				nodeQueue[idx] = v;
			}
		}
	}
}

__global__ void tpe_update(graph_node *tpe, unsigned int *nodeQueue, unsigned int *nodeOffsets, unsigned int *allOffsets, unsigned int *queueCounter, graph_node *edge_list){
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int total_threads = blockDim.x * gridDim.x;
	//listing 2
	int total_edges =  nodeOffsets[queueCounter[0]];
	int loadPerThread;
	if(total_edges % total_threads == 0){
		loadPerThread = nodeOffsets[queueCounter[0]] / total_threads;
	}
	else{
		loadPerThread = nodeOffsets[queueCounter[0]] / total_threads + 1;
	}
	unsigned int tid_begin = thread_id * loadPerThread;
	unsigned int tid_end;
	if(((thread_id + 1)*loadPerThread - 1) > (nodeOffsets[queueCounter[0]] - 1)){
		tid_end = nodeOffsets[queueCounter[0]] - 1;
	}
	else{
		tid_end = (thread_id + 1)*loadPerThread - 1;
	}
	//listing 3
	int i = 0;
	for(int t = 0; t < queueCounter[0]+1; t++){
		if(nodeOffsets[t] > tid_begin){
			if(t != 0){
				i = t-1;
			}
			break;
		}

		if(t == queueCounter[0]){
			i = t;
		}
	}

	int startVertex = nodeQueue[i];
	if(startVertex != -1){
		for(int j = tid_begin; j <= tid_end; j++){
			int phi = j - nodeOffsets[i];
			if(j < nodeOffsets[i+1]){
				uint edgePos = allOffsets[startVertex] + phi;
				tpe[j].src = edge_list[edgePos].src;
				tpe[j].dst = edge_list[edgePos].dst;
				tpe[j].weight = edge_list[edgePos].weight;
			}
			else{
				i++;
				if(i >= queueCounter[0]){
					printf("%d\n", i);
					return;
				}
				startVertex = nodeQueue[i];
				phi = j - nodeOffsets[i];
				uint edgePos = allOffsets[startVertex] + phi;
				tpe[j].src = edge_list[edgePos].src;
				tpe[j].dst = edge_list[edgePos].dst;
				tpe[j].weight = edge_list[edgePos].weight;
			}
			phi++;
		}
	}
}

//device outcore method
void puller_outcore_impl3(std::vector<initial_vertex> * graph, int blockSize, int blockNum, ofstream &outputFile){
	double filter_time = 0;
	double compute_time = 0;

	unsigned int *initDist;
	//set initial distance to max except for source node
	initDist = (unsigned int*)malloc(sizeof(unsigned int)*graph->size());
	initDist[0] = 0;
	for(int i = 1; i < graph->size(); i++){
	    initDist[i] = UINT_MAX;
	}

	unsigned int edge_counter = total_edges_opt(*graph);
	unsigned int initial_edge_counter = edge_counter;

	graph_node *edge_list;
	edge_list = (graph_node*) malloc(sizeof(graph_node)*edge_counter);
	//for each member of edge list set initial values
	unsigned int k = 0;
	for(int i = 0 ; i < graph->size() ; i++){
		std::vector<neighbor> nbrs = (*graph)[i].nbrs;
    for(int j = 0 ; j < nbrs.size() ; j++, k++){
			edge_list[k].src = nbrs[j].srcIndex;
			edge_list[k].dst = i;
			edge_list[k].weight = nbrs[j].edgeValue.weight;
    }
	}

	//sort by source vertex
	//http://www.cplusplus.com/reference/cstdlib/qsort/
	qsort(edge_list, edge_counter, sizeof(graph_node), cmp_edge_src);
	graph_node *device_edge_list;
	hipMalloc((void**)&device_edge_list, sizeof(graph_node)*edge_counter);
	hipMemcpy(device_edge_list, edge_list, sizeof(graph_node)*edge_counter, hipMemcpyHostToDevice);

	//create allneighbor
	unsigned int *allNeighborNumber = new unsigned int[graph->size()];
	for(int i = 0; i < graph->size(); i++){
		allNeighborNumber[i] = 0;
	}
	for(int i = 0; i < graph->size(); i++){
	    std::vector<neighbor> nbrs = (*graph)[i].nbrs;
	    for(int j = 0 ; j < nbrs.size() ; j++){
	    	int src = nbrs[j].srcIndex;
	    	allNeighborNumber[src] += 1;
	    }
	}

  //create allOffsets
  unsigned int *allOffsets = new unsigned int[graph->size() + 1];
  for(int i = 0; i < graph->size(); i++){
	    allOffsets[i] = allNeighborNumber[i];
	}
  thrust::exclusive_scan(allOffsets, allOffsets + graph->size(), allOffsets);
  allOffsets[graph->size()] = allOffsets[graph->size() - 1] + allNeighborNumber[graph->size() - 1];
	unsigned int *device_allOffsets = new unsigned int[graph->size() + 1];
	hipMalloc((void**)&device_allOffsets, sizeof(unsigned int)*(graph->size()+1));
	hipMemcpy(device_allOffsets, allOffsets, sizeof(unsigned int)*(graph->size()+1), hipMemcpyHostToDevice);

	//create a new tpe
	graph_node *tpe;
	tpe = (graph_node*) malloc(sizeof(graph_node)*edge_counter);
	for(int i = 0, j=0 ; i < edge_counter; i++){
		if(edge_list[i].src == 0){
			tpe[j].src = edge_list[i].src;
			tpe[j].dst = edge_list[i].dst;
			tpe[j].weight = edge_list[i].weight;
			j++;
		}
	}

	//create nodeQueue
	unsigned int *nodeQueue = (unsigned int*)malloc(sizeof(unsigned int)*graph->size());
	unsigned int *device_nodeQueue = (unsigned int*)malloc(sizeof(unsigned int)*graph->size());
	hipMalloc((void**)&device_nodeQueue, sizeof(unsigned int)*graph->size());
	for(int i = 0; i < graph->size(); i++){
		nodeQueue[i] = -1;
	}
	hipMemcpy(device_nodeQueue, nodeQueue, sizeof(unsigned int)*graph->size(), hipMemcpyHostToDevice);

	//queueCounter
	unsigned int queueCounter = 0;
	unsigned int *device_queueCounter;
	hipMalloc((void**)&device_queueCounter, sizeof(unsigned int));
	hipMemcpy(device_queueCounter, &queueCounter, sizeof(uint), hipMemcpyHostToDevice);
	hipMemset(device_queueCounter, 0, sizeof(unsigned int));

	unsigned int *distance_cur, *distance_prev;
	graph_node *device_tpe;

	hipMalloc((void**)&device_tpe, sizeof(graph_node)*edge_counter);
	hipMalloc((void**)&distance_cur, sizeof(unsigned int)*graph->size());
	hipMalloc((void**)&distance_prev, sizeof(unsigned int)*graph->size());

	hipMemcpy(distance_cur, initDist, sizeof(unsigned int)*graph->size(), hipMemcpyHostToDevice);
	hipMemcpy(distance_prev, initDist, sizeof(unsigned int)*graph->size(), hipMemcpyHostToDevice);
	hipMemcpy(device_tpe, tpe, sizeof(graph_node)*edge_counter, hipMemcpyHostToDevice);

	for (int i = 0; i < graph->size()-1; ++i) {
		edge_process_opt<<<blockNum, blockSize>>>(device_tpe, initial_edge_counter, distance_cur, distance_prev, device_queueCounter, device_nodeQueue);
		hipDeviceSynchronize();
		hipMemcpy(distance_prev, distance_cur, sizeof(uint)*graph->size(), hipMemcpyDeviceToDevice);
		hipMemcpy(nodeQueue, device_nodeQueue, sizeof(uint)*graph->size(), hipMemcpyDeviceToHost);
		hipMemcpy(&queueCounter, device_queueCounter, sizeof(uint), hipMemcpyDeviceToHost);
		printf("new %d\n",queueCounter);
		if(queueCounter == 0){
			break;
		}
		//create neighborNumber
		unsigned int *neighborNumber = new unsigned int[queueCounter];
		for(int j = 0; j < queueCounter; j++){
			int idx = nodeQueue[j];
			neighborNumber[j] = allNeighborNumber[idx];
		}
		//create nodeOffset
		unsigned int *nodeOffsets = new unsigned int[queueCounter + 1];
		for(int i = 0; i < queueCounter; i++){
		    nodeOffsets[i] = neighborNumber[i];
		}
		thrust::exclusive_scan(nodeOffsets, nodeOffsets + queueCounter, nodeOffsets);
		nodeOffsets[queueCounter] = nodeOffsets[queueCounter - 1] + neighborNumber[queueCounter - 1];
		unsigned int *device_nodeOffsets = new unsigned int[queueCounter + 1];
		hipMalloc((void**)&device_nodeOffsets, sizeof(uint)*(queueCounter + 1));
		hipMemcpy(device_nodeOffsets, nodeOffsets, sizeof(uint)*(queueCounter + 1), hipMemcpyHostToDevice);

		free(tpe);
		hipFree(device_tpe);
		graph_node *tpe = (graph_node*) malloc(sizeof(graph_node)*edge_counter);
		graph_node *device_tpe;
		hipMalloc((void**)&device_tpe, sizeof(graph_node)*edge_counter);
		hipMemcpy(device_tpe, tpe, sizeof(graph_node)*edge_counter, hipMemcpyHostToDevice);

		tpe_update<<<blockNum, blockSize>>>(device_tpe, device_nodeQueue, device_nodeOffsets, device_allOffsets, device_queueCounter, device_edge_list);
		hipDeviceSynchronize();
		hipMemcpy(tpe, device_tpe, sizeof(graph_node)*edge_counter, hipMemcpyDeviceToHost);
		hipMemcpy(device_tpe, tpe, sizeof(graph_node)*edge_counter, hipMemcpyHostToDevice);
		hipMemset(device_queueCounter, 0, sizeof(unsigned int));
		hipMemcpy(&queueCounter, device_queueCounter, sizeof(uint), hipMemcpyDeviceToHost);
	}

	hipMemcpy(initDist, distance_cur, sizeof(uint)*graph->size(), hipMemcpyDeviceToHost);

	for(int i=0; i < graph->size(); i++){
		if(initDist[i] == UINT_MAX){
		    outputFile << i << ":" << "INF" << endl;
		}
		else{
		    outputFile << i << ":" << initDist[i] << endl;
		}
	}
	//http://www.geeksforgeeks.org/g-fact-30/
	free(initDist);
	hipFree(distance_cur);
	hipFree(distance_prev);
	free(tpe);
	//free(nodeQueue);
	// free(allOffsets);
	// free(initDist);
	// cout<<"test";
	// hipFree(distance_cur);
	// hipFree(distance_prev);
	// hipFree(device_tpe);
	// hipFree(device_nodeQueue);
}

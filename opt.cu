#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <thrust/scan.h>
#include "utils.h"
#include "cuda_error_check.cuh"
#include "initial_graph.hpp"
#include "parse_graph.hpp"

using namespace std;

//comparator function used by qsort
int cmp_edge_src(const void *a, const void *b){
	return ( (((graph_node *)a)->src) - (((graph_node *)b)->src));
}

//get total edges
unsigned int total_edges_opt(std::vector<initial_vertex>& graph){
	unsigned int edge_counter = 0;
	for(int i = 0 ; i < graph.size() ; i++){
	    edge_counter += graph[i].nbrs.size();
	}
	return edge_counter;
}

//outcore
//kernel outcore method
__global__ void edge_process_opt(graph_node *L, const uint edge_counter, unsigned int *distance_cur, unsigned int *distance_prev, unsigned int *queueCounter, unsigned int *nodeQueue){

	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int total_threads = blockDim.x * gridDim.x;
	int warp_id = thread_id/32;
	int warp_num;
	if(total_threads % 32 == 0){
		warp_num = total_threads/32;
	}
	else{
		warp_num = total_threads/32 + 1;
	}
	int lane_id = thread_id % 32;

	//given in the psuedocode
	int load = (edge_counter % warp_num == 0) ? edge_counter/warp_num : edge_counter/warp_num+1;
	int beg = load * warp_id;
	int end = beg + load;
	if(edge_counter < beg + load)
		end = edge_counter;
	beg = beg + lane_id;

	unsigned int u, v, w;
	graph_node *edge;
	//printf("Hello thread %d\n", threadIdx.x);
	for(int i = beg; i < end; i+=32){
		edge = L + i;
		u = edge->src;
		v = edge->dst;
		w = edge->weight;
		if(distance_prev[u] != UINT_MAX && distance_prev[u] + w < distance_prev[v] && distance_prev[u] + w < distance_cur[v]){
			printf("Hello thread %d\n", threadIdx.x);
			int old_val = atomicMin(&distance_cur[v], distance_prev[u] + w);
			printf("old_val %d\n", old_val);
			if(old_val >= distance_prev[v] && distance_prev[u] + w < old_val){
				int idx = atomicAdd(&queueCounter[0], 1);
				printf("idx %d\n", idx);
				nodeQueue[idx] = v;
				printf("v %d\n", v);
			}
		}
	}
}

//device outcore method
void puller_outcore_impl3(std::vector<initial_vertex> * graph, int blockSize, int blockNum, ofstream &outputFile){
	double filter_time = 0;
	double compute_time = 0;

	unsigned int *initDist;
	//set initial distance to max except for source node
	initDist = (unsigned int*)malloc(sizeof(unsigned int)*graph->size());
	initDist[0] = 0;
	for(int i = 1; i < graph->size(); i++){
	    initDist[i] = UINT_MAX;
	}

	unsigned int edge_counter = total_edges_opt(*graph);
	unsigned int initial_edge_counter = edge_counter;

	graph_node *edge_list;
	edge_list = (graph_node*) malloc(sizeof(graph_node)*edge_counter);
	//for each member of edge list set initial values
	unsigned int k = 0;
	for(int i = 0 ; i < graph->size() ; i++){
		std::vector<neighbor> nbrs = (*graph)[i].nbrs;
    for(int j = 0 ; j < nbrs.size() ; j++, k++){
			edge_list[k].src = nbrs[j].srcIndex;
			edge_list[k].dst = i;
			edge_list[k].weight = nbrs[j].edgeValue.weight;
    }
	}

	//sort by source vertex
	//http://www.cplusplus.com/reference/cstdlib/qsort/
	qsort(edge_list, edge_counter, sizeof(graph_node), cmp_edge_src);

	//create allneighbor
	unsigned int *allNeighborNumber = new unsigned int[graph->size()];
	for(int i = 0; i < graph->size(); i++){
		allNeighborNumber[i] = 0;
	}
	for(int i = 0; i < graph->size(); i++){
	    std::vector<neighbor> nbrs = (*graph)[i].nbrs;
	    for(int j = 0 ; j < nbrs.size() ; j++){
	    	int src = nbrs[j].srcIndex;
	    	allNeighborNumber[src] += 1;
	    }
	}

  //create allOffsets
  unsigned int *allOffsets = new unsigned int[graph->size() + 1];
  for(int i = 0; i < graph->size(); i++){
	    allOffsets[i] = allNeighborNumber[i];
	}
  thrust::exclusive_scan(allOffsets, allOffsets + graph->size(), allOffsets);
  allOffsets[graph->size()] = allOffsets[graph->size() - 1] + allNeighborNumber[graph->size() - 1];

	std::cout << "before l_new" << "\n";
	//create a new l'
	graph_node *L_new;
	L_new = (graph_node*) malloc(sizeof(graph_node)*edge_counter);
	for(int i = 0, j=0 ; i < edge_counter; i++){
		if(edge_list[i].src == 0){
			L_new[j].src = edge_list[i].src;
			L_new[j].dst = edge_list[i].dst;
			L_new[j].weight = edge_list[i].weight;
			j++;
		}
	}

	std::cout << "before nodeQueue" << "\n";
	//create nodeQueue
	unsigned int *nodeQueue = (unsigned int*)malloc(sizeof(unsigned int)*graph->size());
	unsigned int *device_nodeQueue = (unsigned int*)malloc(sizeof(unsigned int)*graph->size());
	hipMalloc((void**)&device_nodeQueue, sizeof(unsigned int)*graph->size());
	for(int i = 0; i < graph->size(); i++){
		nodeQueue[i] = -1;
	}
	hipMemcpy(device_nodeQueue, nodeQueue, sizeof(unsigned int)*graph->size(), hipMemcpyHostToDevice);

	std::cout << "before queueCounter" << "\n";
	//queueCounter
	unsigned int *queueCounter = 0;
	unsigned int *device_queueCounter;
	hipMalloc((void**)&device_queueCounter, sizeof(unsigned int));
	hipMemcpy(device_queueCounter, &queueCounter, sizeof(uint), hipMemcpyHostToDevice);
	hipMemset(device_queueCounter, 0, sizeof(unsigned int));

	unsigned int *distance_cur, *distance_prev, *temp_distance, *device_warp_update_ds, *device_edge_counter;
	graph_node *L;
	graph_node *device_L_new;

	int total_threads = blockSize * blockNum;
	int warp_num;
	if(total_threads % 32 == 0){
		warp_num = total_threads/32;
	}
	else{
		warp_num = total_threads/32 + 1;
	}

	std::cout << "before cuda thingys" << "\n";
	unsigned int *temp_warp_update = new unsigned int[warp_num];

	hipMalloc((void**)&device_edge_counter, sizeof(unsigned int));
	hipMalloc((void**)&temp_distance, sizeof(unsigned int)*graph->size());

	hipMalloc((void**)&device_warp_update_ds, sizeof(unsigned int)*warp_num);
	hipMalloc((void**)&L, sizeof(graph_node)*edge_counter);
	hipMalloc((void**)&device_L_new, sizeof(graph_node)*edge_counter);
	hipMalloc((void**)&distance_cur, sizeof(unsigned int)*graph->size());
	hipMalloc((void**)&distance_prev, sizeof(unsigned int)*graph->size());

	hipMemcpy(distance_cur, initDist, sizeof(unsigned int)*graph->size(), hipMemcpyHostToDevice);
	hipMemcpy(distance_prev, initDist, sizeof(unsigned int)*graph->size(), hipMemcpyHostToDevice);
	hipMemcpy(L, edge_list, sizeof(graph_node)*edge_counter, hipMemcpyHostToDevice);
	hipMemcpy(device_L_new, L_new, sizeof(graph_node)*edge_counter, hipMemcpyHostToDevice);
	hipMemcpy(device_edge_counter, &edge_counter, sizeof(uint), hipMemcpyHostToDevice);

	for (int i = 0; i < graph->size()-1; ++i) {
		std::cout << "inside for" << "\n";
		setTime();

		hipMemset(device_warp_update_ds, 0, sizeof(uint)*warp_num);
		hipMemcpy(temp_distance, distance_cur, sizeof(uint)*graph->size(), hipMemcpyDeviceToDevice);
		std::cout << "before kernel" << "\n";
		edge_process_opt<<<blockNum, blockSize>>>(device_L_new, initial_edge_counter, distance_cur, distance_prev, device_queueCounter, device_nodeQueue);
		hipDeviceSynchronize();
		std::cout << "after kernel"<< "\n";
		hipMemcpy(distance_prev, distance_cur, sizeof(uint)*graph->size(), hipMemcpyDeviceToDevice);
		hipMemcpy(nodeQueue, device_nodeQueue, sizeof(uint)*graph->size(), hipMemcpyDeviceToHost);

		for(int j = 0; j < 10; j++){
			std::cout << nodeQueue[j] << "\n";
		}
		break;
		compute_time += getTime();

		setTime();

		filter_time += getTime();
	}

	std::cout << "Compute Time: " << compute_time << "\n";
	std::cout << "Filter Time: " << filter_time;

	hipMemcpy(initDist, distance_cur, sizeof(uint)*graph->size(), hipMemcpyDeviceToHost);

	for(int i=0; i < graph->size(); i++){
		if(initDist[i] == UINT_MAX){
		    outputFile << i << ":" << "INF" << endl;
		}
		else{
		    outputFile << i << ":" << initDist[i] << endl;
		}
	}
	free(initDist);
	hipFree(L);
	hipFree(device_edge_counter);
	hipFree(distance_cur);
	hipFree(distance_prev);
	hipFree(temp_distance);
	hipFree(device_warp_update_ds);
}
